
#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>
#include <hipfft/hipfft.h>

 int main()
 {
    const int N = 1024;

    std::vector<float>        A(N);
    std::vector<hipfftComplex> C(N);

    std::mt19937 mersenne_engine{42};  // Generates random integers
    std::uniform_real_distribution<float> dist{-0.1f, 0.1f};

    auto gen = [&dist, &mersenne_engine](){ return dist(mersenne_engine); };
    generate(A.begin(), A.end(), gen);
    std::fill(C.begin(), C.end(), hipfftComplex{0.0f, 0.0f});

    hipError_t cudaStatus = hipSuccess;

    // Create and set stream, create events:
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if( cudaStatus != hipSuccess ){ std::cout << "Error creating CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    hipEvent_t evt[3];
    for(auto& e : evt)
    {
        auto cudaStatus = hipEventCreate(&e);
        if(cudaStatus != hipSuccess){ std::cout << "Error creating event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    hipfftResult result = HIPFFT_SUCCESS;

    // Create FFT Plans:
    hipfftHandle   plan_fwd, plan_bwd;
    result = hipfftPlan1d(&plan_fwd, N, HIPFFT_R2C, 1);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot create fwd cufft plan: " << result << "\n"; return -1; }

    result = hipfftSetStream(plan_fwd, stream);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot set stream for fwd cufft: " << result << "\n"; return -1; }

    result = hipfftPlan1d(&plan_bwd, N/2+1, HIPFFT_C2C, 1);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot create bwd cufft plan: " << result << "\n"; return -1; }

    result = hipfftSetStream(plan_bwd, stream);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot set stream for bwd cufft: " << result << "\n"; return -1; }
    

    // Allocate device data:
    float*        bA = nullptr;
    hipfftComplex* bB = nullptr;
    hipfftComplex* bC = nullptr;
    
    cudaStatus = hipMalloc((void**)&bA, N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for float array A:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&bB, sizeof(hipfftComplex)*(N/2+1));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for complex array B:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&bC, sizeof(hipfftComplex)*(N/2+1));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for complex array C:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMemcpy(bA, A.data(), N*sizeof(float), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){ std::cout << "Cannot copy data to device memory:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[0], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (0): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    result = hipfftExecR2C(plan_fwd, (hipfftReal*)bA, bB);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot execute R2C fft: " << result << "\n"; return -1; }

    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess ){ std::cout << "CUDA error in kernel call R2C: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[1], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (1): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    result = hipfftExecC2C(plan_bwd, bB, bC, HIPFFT_BACKWARD);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot execute inverse C2C fft: " << result << "\n"; return -1; }

    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess ){ std::cout << "CUDA error in kernel call R2C: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[2], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (2): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    hipDeviceSynchronize();

    float dt_fwd = 0.0f, dt_bwd; //milliseconds
    cudaStatus = hipEventElapsedTime(&dt_fwd, evt[0], evt[1]);
    if(cudaStatus != hipSuccess){ std::cout << "Error getting event 0-1 elapsed time: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    cudaStatus = hipEventElapsedTime(&dt_bwd, evt[1], evt[2]);
    if(cudaStatus != hipSuccess){ std::cout << "Error getting event 1-2 elapsed time: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    std::cout << "Forward FFT took: " << dt_fwd << " ms.\n";
    std::cout << "Backward FFT took: " << dt_bwd << " ms.\n";

    // Copy device memory to host
    cudaStatus = hipMemcpy(C.data(), bC, sizeof(hipfftComplex)*(N/2+1), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){ std::cout << "Error copying data back: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(bA);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for array A: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(bB);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for array B: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(bC);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for array C: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    for(auto& e : evt)
    {
        cudaStatus = hipEventDestroy(e);
        if(cudaStatus != hipSuccess){ std::cout << "Error destroying event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    result = hipfftSetStream(plan_fwd, 0);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot reset cufft stream: " << result << "\n"; return -1; }
    result = hipfftSetStream(plan_bwd, 0);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot reset cufft stream: " << result << "\n"; return -1; }

    cudaStatus = hipStreamDestroy(stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error destroying CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    result = hipfftDestroy(plan_fwd);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot destroy cufft plan: " << result << "\n"; return -1; }

    result = hipfftDestroy(plan_bwd);
    if(result != HIPFFT_SUCCESS){ std::cout << "Cannot destroy cufft plan: " << result << "\n"; return -1; }

    for(int i=0; i<(int)C.size(); ++i)
    {
        std::cout << i << "   " << A[i] << "   " << C[i].x << "   " << C[i].y << "\n";
    }

    return 0;
  }
#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>
#include <hipblas.h>
#include "cpu_matmul.h"

int main()
{
    const int N = 1024;

    std::vector<float> A(N*N);
    std::vector<float> B(N*N);
    std::vector<float> C1(N*N);
    std::vector<float> C2(N*N);

    std::mt19937 mersenne_engine{42};  // Generates random integers
    std::uniform_real_distribution<float> dist{-0.1f, 0.1f};

    auto gen = [&dist, &mersenne_engine](){ return dist(mersenne_engine); };
    generate(A.begin(), A.end(), gen);
    generate(B.begin(), B.end(), gen);
    std::fill(C1.begin(), C1.end(), 0.0f);
    std::fill(C2.begin(), C2.end(), 0.0f);

    // Initialize cublas:
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    status = hipblasCreate(&handle);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot initialize cublas:" << status << "\n"; return -1; }

    hipError_t cudaStatus = hipSuccess;

    // Create and set stream, create events:
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if( cudaStatus != hipSuccess ){ std::cout << "Error creating CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    status = hipblasSetStream(handle, stream);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot set cublas stream:" << status << "\n"; return -1; }

    hipEvent_t evt[2];
    for(auto& e : evt)
    {
        auto cudaStatus = hipEventCreate(&e);
        if(cudaStatus != hipSuccess){ std::cout << "Error creating event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    // Allocate device data:
    float* cbA = nullptr;
    float* cbB = nullptr;
    float* cbC = nullptr;

    cudaStatus = hipMalloc((void**)&cbA, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix A:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&cbB, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix B:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&cbC, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix C:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    // Set matrix data:
    status = hipblasSetMatrix(N, N, sizeof(float), A.data(), N, cbA, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot upload contents for cublas matrix A:" << status << "\n"; return -1; }

    status = hipblasSetMatrix(N, N, sizeof(float), B.data(), N, cbB, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot upload contents for cublas matrix B:" << status << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[0], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (0): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    // Matrix multiplication funciton: https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
    // All matrixes need to be transposed, including the result C, but since C^T = (AB)^T = (B^T A^T), we dont need to transpose the input matrices, just change their order:
    float one = 1.0f;
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &one, cbB, N, cbA, N, &one, cbC, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot start cublas matrix multiplication:" << status << "\n"; return -1; }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess ){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[1], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (1): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    hipDeviceSynchronize();

    float dt = 0.0f; //milliseconds
    cudaStatus = hipEventElapsedTime(&dt, evt[0], evt[1]);
    if(cudaStatus != hipSuccess){ std::cout << "Error getting event 0-1 elapsed time: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    
    status = hipblasGetMatrix(N, N, sizeof(float), cbC, N, C2.data(), N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot copy back results for cublas matrix C:" << status << "\n"; return -1; }

    cudaStatus = hipFree(cbA);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix A:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(cbB);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix B:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(cbC);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix C:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    for(auto& e : evt)
    {
        cudaStatus = hipEventDestroy(e);
        if(cudaStatus != hipSuccess){ std::cout << "Error destroying event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    status = hipblasSetStream(handle, 0);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot reset cublas stream:" << status << "\n"; return -1; }

    cudaStatus = hipStreamDestroy(stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error destroying CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    status = hipblasDestroy(handle);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Error shutting down cublas:" << status << "\n"; return -1; }

    auto t1 = std::chrono::high_resolution_clock::now();
    cpu_matmul_improved(C1, A, B, N);
    auto t2 = std::chrono::high_resolution_clock::now();

    const float max_err = 1e-5f;
    auto comparator = [max_err](float l, float r){ return std::abs(l-r) < max_err; };
    
    for(int i=0; i<N*N; ++i)
	{
        if( !comparator(C1[i], C2[i]) )
        {
            std::cout << "C1 vs C2 [" << i << "] : " << C1[i] << "   " << C2[i] << " absolute error: " << std::abs(C1[i]-C2[i]) << "\n";
        }
    }

    if( std::equal(C1.begin(), C1.end(), C2.begin(), comparator) )
	{
		std::cout << "GPU improved matches CPU naive.\n";
	}
	else
	{
		std::cout << "Mismatch in CPU and GPU results.\n";
	}
    
    std::cout << "CPU improved Computation took:     " << std::chrono::duration_cast<std::chrono::microseconds>(t2-t1).count()/1000.0f << " ms\n";
    std::cout << "Cublas matrix multiplication took: " << dt << " ms.\n";

    return 0;
}
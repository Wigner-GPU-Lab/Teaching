
#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>
#include <hipblas.h>

 int main()
 {
    const int N = 1024;

    std::vector<float> A(N*N);
    std::vector<float> B(N*N);
    std::vector<float> C1(N*N);
    std::vector<float> C2(N*N);

    std::mt19937 mersenne_engine{42};  // Generates random integers
    std::uniform_real_distribution<float> dist{-0.1f, 0.1f};

    auto gen = [&dist, &mersenne_engine](){ return dist(mersenne_engine); };
    generate(A.begin(), A.end(), gen);
    generate(B.begin(), B.end(), gen);
    std::fill(C1.begin(), C1.end(), 0.0f);
    std::fill(C2.begin(), C2.end(), 0.0f);

    // Initialize cublas:
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    status = hipblasCreate(&handle);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot initialize cublas:" << status << "\n"; return -1; }

    hipError_t cudaStatus = hipSuccess;

    // Create and set stream, create events:
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if( cudaStatus != hipSuccess ){ std::cout << "Error creating CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    status = hipblasSetStream(handle, stream);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot set cublas stream:" << status << "\n"; return -1; }

    hipEvent_t evt[2];
    for(auto& e : evt)
    {
        auto cudaStatus = hipEventCreate(&e);
        if(cudaStatus != hipSuccess){ std::cout << "Error creating event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    // Allocate device data:
    float* cbA = nullptr;
    float* cbB = nullptr;
    float* cbC = nullptr;
    
    cudaStatus = hipMalloc((void**)&cbA, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix A:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&cbB, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix B:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipMalloc((void**)&cbC, N*N*sizeof(float));
    if(cudaStatus != hipSuccess){ std::cout << "Cannot allocate device memory for matrix C:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    // Set matrix data:
    status = hipblasSetMatrix(N, N, sizeof(float), A.data(), N, cbA, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot upload contents for cublas matrix A:" << status << "\n"; return -1; }

    status = hipblasSetMatrix(N, N, sizeof(float), B.data(), N, cbB, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot upload contents for cublas matrix B:" << status << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[0], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (0): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    // Matrix multiplication funciton: https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
    float one = 1.0f;
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &one, cbA, N, cbB, N, &one, cbC, N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot start cublas matrix multiplication:" << status << "\n"; return -1; }

    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess ){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipEventRecord(evt[1], stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error recording event (1): " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    hipDeviceSynchronize();

    float dt = 0.0f; //milliseconds
    cudaStatus = hipEventElapsedTime(&dt, evt[0], evt[1]);
    if(cudaStatus != hipSuccess){ std::cout << "Error getting event 0-1 elapsed time: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    std::cout << "Cublas matrix multiplication took " << dt << " ms.\n";

    status = hipblasGetMatrix(N, N, sizeof(float), cbC, N, C2.data(), N);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot copy back results for cublas matrix C:" << status << "\n"; return -1; }

    cudaStatus = hipFree(cbA);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix A:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(cbB);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix B:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    cudaStatus = hipFree(cbC);
    if(cudaStatus != hipSuccess){ std::cout << "Error freeing device memory for matrix C:" << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    for(auto& e : evt)
    {
        cudaStatus = hipEventDestroy(e);
        if(cudaStatus != hipSuccess){ std::cout << "Error destroying event: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }
    }

    status = hipblasSetStream(handle, 0);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Cannot reset cublas stream:" << status << "\n"; return -1; }

    cudaStatus = hipStreamDestroy(stream);
    if(cudaStatus != hipSuccess){ std::cout << "Error destroying CUDA stream: " << hipGetErrorString(cudaStatus) << "\n"; return -1; }

    status = hipblasDestroy(handle);
    if(status != HIPBLAS_STATUS_SUCCESS){ std::cout << "Error shutting down cublas:" << status << "\n"; return -1; }

    return 0;
  }
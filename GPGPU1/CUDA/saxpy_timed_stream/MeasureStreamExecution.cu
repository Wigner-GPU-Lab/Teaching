
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <chrono>

__global__ void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        y[i] = a*x[i] + y[i];
    }
}

int main()
{
    size_t sz = 1 << 24;
    std::cout << "Data size is: " << sz << "\n";
    const float A = 2.0f;

	std::vector<float> X(sz);
	std::vector<float> Y(sz);
    std::vector<float> R(sz);
    
    std::generate(X.begin(), X.end(), [d=0.f, dd=+1.0f/sz]()mutable{ d += dd; return d; });
    std::generate(Y.begin(), Y.end(), [d=1.f, dd=-1.0f/sz]()mutable{ d += dd; return d; });
	
	float* devX = nullptr;
	float* devY = nullptr;

    hipError_t err = hipSuccess;

    hipStream_t stream;
    err = hipStreamCreate(&stream);
    if( err != hipSuccess ){ std::cout << "Error creating CUDA stream: " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&devX, sz*sizeof(float) );
	if( err != hipSuccess ){ std::cout << "Error allocating CUDA memory (X): " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&devY, sz*sizeof(float) );
	if( err != hipSuccess ){ std::cout << "Error allocating CUDA memory (Y): " << hipGetErrorString(err) << "\n"; return -1; }
	
	//err = cudaMemcpy( devX, X.data(), sz*sizeof(float), cudaMemcpyHostToDevice );
    err = hipMemcpyAsync(devX, X.data(), sz*sizeof(float), hipMemcpyHostToDevice, stream);
    if( err != hipSuccess ){ std::cout << "Error copying memory to device (X): " << hipGetErrorString(err) << "\n"; return -1; }
    
    //err = cudaMemcpy( devY, Y.data(), sz*sizeof(float), cudaMemcpyHostToDevice );
    err = hipMemcpyAsync(devY, Y.data(), sz*sizeof(float), hipMemcpyHostToDevice, stream);
    if( err != hipSuccess ){ std::cout << "Error copying memory to device (Y): " << hipGetErrorString(err) << "\n"; return -1; }
    
    hipEvent_t evt[3];
    for(auto& e : evt)
    {
        err = hipEventCreate(&e);
        if( err != hipSuccess ){ std::cout << "Error creating event: " << hipGetErrorString(err) << "\n"; return -1; }
    }

	dim3 dimGrid( sz/512 );
    dim3 dimBlock( 512 );
    
    err = hipEventRecord(evt[0], stream);
    if( err != hipSuccess ){ std::cout << "Error recording event (0): " << hipGetErrorString(err) << "\n"; return -1; }
    
    saxpy<<<dimGrid, dimBlock, 0, stream>>>((int)sz, A, devX, devY);
    
    err = hipGetLastError();
	if (err != hipSuccess ){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
    
    err = hipEventRecord(evt[1], stream);
    if( err != hipSuccess ){ std::cout << "Error recording event (1): " << hipGetErrorString(err) << "\n"; return -1; }

	//err = cudaMemcpy( R.data(), devY, sz*sizeof(float), cudaMemcpyDeviceToHost );
    err = hipMemcpyAsync( R.data(), devY, sz*sizeof(float), hipMemcpyDeviceToHost, stream );
	if( err != hipSuccess ){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipEventRecord(evt[2], stream);
    if( err != hipSuccess ){ std::cout << "Error recording event (2): " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipEventSynchronize(evt[2]);
    if( err != hipSuccess ){ std::cout << "Error during synchronize with event: " << hipGetErrorString(err) << "\n"; return -1; }

    float dt1 = 0.0f, dt2 = 0.0f;//milliseconds
    err = hipEventElapsedTime(&dt1, evt[0], evt[1]);
    if( err != hipSuccess ){ std::cout << "Error getting event 0-1 elapsed time: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipEventElapsedTime(&dt2, evt[1], evt[2]);
    if( err != hipSuccess ){ std::cout << "Error getting event 1-2 elapsed time: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree( devX );
    if( err != hipSuccess ){ std::cout << "Error freeing allocation (X): " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree( devY );
    if( err != hipSuccess ){ std::cout << "Error freeing allocation (Y): " << hipGetErrorString(err) << "\n"; return -1; }

    for(auto& e : evt)
    {
        err = hipEventDestroy(e);
        if( err != hipSuccess ){ std::cout << "Error destroying event: " << hipGetErrorString(err) << "\n"; return -1; }
    }

    err = hipStreamDestroy(stream);
    if( err != hipSuccess ){ std::cout << "Error destroying CUDA stream: " << hipGetErrorString(err) << "\n"; return -1; }

    // skip printing:
	/*for( auto r : R )
	{
		std::cout << r << "\n";
	}*/

    auto t0 = std::chrono::high_resolution_clock::now();
	std::transform(X.begin(), X.end(), Y.begin(), Y.begin(), [a = A](float x, float y){ return a*x + y; });
    auto t1 = std::chrono::high_resolution_clock::now();
    
	if(std::equal(R.begin(), R.end(), Y.begin()))
	{
		std::cout << "Success\n";
	}
    else{ std::cout << "Mismatch between CPU and GPU results.\n"; }
    
    std::cout << "CPU computation took:         " << std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count()/1000.0f << " ms\n";
    std::cout << "GPU computation took:         " << dt1 << " ms.\n";
    std::cout << "GPU device-to-host copy took: " << dt2 << " ms.\n";

	return 0;
}


#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>

struct v3{ float x, y, z; };
struct v4{ float x, y, z, m; };

__host__ __device__ float sq(float x){ return x*x; }
__host__ __device__ float cube(float x){ return x*x*x; }

void cpu_nbody_opt(std::vector<v3>& F, std::vector<v4> const& P, float G, float eps) 
{
    int N = (int)P.size();
    for(int i=0; i<N; ++i)
    {
        v3 sum = {0.0f, 0.0f, 0.0f};
        float x = P[i].x;
        float y = P[i].y;
        float z = P[i].z;
        for(int j=0; j<N; ++j)
        {
            if(i == j){ continue; }
            float dx = P[j].x - x;
            float dy = P[j].y - y;
            float dz = P[j].z - z;
            float rec = P[j].m / (cube( sqrt(sq(dx) + sq(dy) + sq(dz)) ));
            sum.x += dx * rec;
            sum.y += dy * rec;
            sum.z += dz * rec;
        }
        float gm = -G * P[i].m;
        F[i] =  v3{gm * sum.x, gm * sum.y, gm * sum.z};
    }
}
static const int blockSize = 1024;
__global__ void gpu_nbody_opt(float3* F, float4* P, float G, float eps) 
{
    //blockDim.x == blockSize is used below:
    unsigned int N = blockSize*gridDim.x;
    unsigned int i = blockIdx.x*blockSize + threadIdx.x;
    __shared__ float4 parts[blockSize];

    float3 sum = {0.0f, 0.0f, 0.0f};
    float4 Pi = P[i];
    float x = Pi.x;
    float y = Pi.y;
    float z = Pi.z;
    for(int J=0; J<N; J+= blockSize)
    {
        parts[threadIdx.x] = P[J+threadIdx.x];
        __syncthreads();
        //instead of checking i != J+j, we check: j != i-J, thus we can precompute

        int iminusJ = (int)i - J;
        for(int j=0; j<blockSize; ++j)
        {
            float4 Pj = parts[j];
            float dx = Pj.x - x;
            float dy = Pj.y - y;
            float dz = Pj.z - z;
            float rsqrt = iminusJ != j ? rsqrtf( sq(dx) + sq(dy) + sq(dz) ) : 0.0f;
            float rec = Pj.w * cube(rsqrt);
            //float rec = Pj.w  * cube( rnorm3df(dx, dy, dz) );
            sum.x += dx * rec;
            sum.y += dy * rec;
            sum.z += dz * rec;
        }
        __syncthreads();
    }
    float gm = -G * Pi.w;
    F[i] = float3{gm*sum.x, gm*sum.y, gm*sum.z};
}

int main()
{
    const int N = 8192*4;
    const int block_sz = blockSize;
    const int n_blocks = N / block_sz;
    const float G = 1e-2;
    const float eps = 1e-9;

    std::vector<v4> Points(N);
    std::vector<v3> Forces1(N), Forces2(N);

    std::mt19937 mersenne_engine{42};  // Generates random integers
    std::uniform_real_distribution<float> dist{-0.1f, 0.1f};

    auto gen = [&dist, &mersenne_engine](){ return v4{dist(mersenne_engine), dist(mersenne_engine), dist(mersenne_engine), dist(mersenne_engine)+0.3f}; };
    generate(Points.begin(), Points.end(), gen);
    std::fill(Forces1.begin(), Forces1.end(), v3{0.0f, 0.0f, 0.0f});
    std::fill(Forces2.begin(), Forces2.end(), v3{0.0f, 0.0f, 0.0f});
	
	float* pP = nullptr;
    float* pF = nullptr;

    hipEvent_t evt[2];
    for(auto& e : evt){ hipEventCreate(&e); }

	hipError_t err = hipSuccess;
	err = hipMalloc( (void**)&pP, N*sizeof(v4) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&pF, N*sizeof(v3) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMemcpy( pP, Points.data(), N*sizeof(v4), hipMemcpyHostToDevice );
    if( err != hipSuccess){ std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }
    
    {
        dim3 dimGrid( n_blocks );
        dim3 dimBlock( block_sz );
        hipEventRecord(evt[0]);
        gpu_nbody_opt<<<dimGrid, dimBlock>>>((float3*)pF, (float4*)pP, G, eps);
        err = hipGetLastError();
	    if (err != hipSuccess){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
        hipEventRecord(evt[1]);
    }

	err = hipMemcpy( Forces2.data(), pF, N*sizeof(v3), hipMemcpyDeviceToHost );
	if( err != hipSuccess){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( pP );
	if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( pF );
	if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    hipEventSynchronize(evt[1]);
    float dt = 0.0f;//milliseconds
    hipEventElapsedTime(&dt, evt[0], evt[1]);

    for(auto& e : evt){ hipEventDestroy(e); }

    auto t0 = std::chrono::high_resolution_clock::now();
    cpu_nbody_opt(Forces1, Points, G, eps);
    auto t1 = std::chrono::high_resolution_clock::now();

    bool mismatch = false;
    const float max_err = 1e-2f;
    for(int i=0; i<N; ++i)
	{
        v3 f1 = Forces1[i];
        v3 f2 = Forces2[i];
        float length_error = std::sqrt((f1.x-f2.x)*(f1.x-f2.x) + (f1.y-f2.y)*(f1.y-f2.y) + (f1.z-f2.z)*(f1.z-f2.z));
        if( length_error > max_err )
        {
            mismatch = true;
            std::cout << "[" << i << "] : " << f1.x << "   " << f2.x << ",   " << f1.y << "   " << f2.y << ",   " << f1.z << "   " << f2.z << " difference length: " << length_error << "\n";
        }
    }

    if( !mismatch )
	{
		std::cout << "Success.\n";
	}
	else
	{
		std::cout << "Mismatch in CPU and GPU result.\n";
	}
    
    std::cout << "CPU Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count()/1000.0f << " ms\n";
    std::cout << "GPU Computation took: " << dt << " ms.\n";
	return 0;
}

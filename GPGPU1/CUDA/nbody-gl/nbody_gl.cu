#include "hip/hip_runtime.h"
#ifdef _WIN32
	#define WIN32_LEAN_AND_MEAN 1
	#include <windows.h>
#else
#endif

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#ifdef _WIN32
#else
	#include <GL/glx.h>
	#include <GL/glext.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>

#include <string>
#include <algorithm>
#include <vector>
#include <random>
#include <iostream>

__host__ __device__ float sq(float x){ return x*x; }
__host__ __device__ float cube(float x){ return x*x*x; }

__global__ void gpu_nbody_opt(float3* V, float4* P, float G, float dt) 
{
    unsigned int N = blockDim.x*gridDim.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    float3 sum = {0.0f, 0.0f, 0.0f};
    float4 Pi = P[i];
    float x = Pi.x;
    float y = Pi.y;
	float z = Pi.z;
	
	for(int j=0; j<N; ++j)
    {
        float4 Pj = P[j];
        float dx = Pj.x - x;
        float dy = Pj.y - y;
        float dz = Pj.z - z;
        float rsqrt = i != j ? rsqrtf( sq(dx) + sq(dy) + sq(dz) ) : 0.0f;
        float rec = Pj.w * cube(rsqrt);
        sum.x += dx * rec;
        sum.y += dy * rec;
        sum.z += dz * rec;
    }
    
    //a = F/m = -G * sum
    //pos = pos + vel * dt + acc / 2 *dt^2
    float3 Vi = V[i];
    float vx = Vi.x + G*sum.x * dt;
    float vy = Vi.y + G*sum.y * dt;
    float vz = Vi.z + G*sum.z * dt;

    x = x + (Vi.x + G/2*sum.x * dt)*dt;
    y = y + (Vi.y + G/2*sum.y * dt)*dt;
    z = z + (Vi.z + G/2*sum.z * dt)*dt;

    P[i] = float4{x, y, z, Pi.w};
    V[i] = float3{vx, vy, vz};

}

struct Velocity{ float x, y, z; };
struct Particle{ float x, y, z, m; };

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	template<typename fptr_type>
	fptr_type load_extension_pointer(const char* name){ return reinterpret_cast<fptr_type>(wglGetProcAddress(name)); }
#else
	template<typename fptr_type>
	fptr_type load_extension_pointer(const char* name){ return reinterpret_cast<fptr_type>(glXGetProcAddressARB((const GLubyte*)name)); }
#endif

static void error_callback(int error, const char* description)
{
    std::cout << "Error: " << description << "\n";
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
	{
        glfwSetWindowShouldClose(window, GLFW_TRUE);
	}
}

static inline const char* glErrorToString(GLenum err)
{
#define CASE_RETURN_MACRO(arg) case arg: return #arg
    switch(err)
    {
        CASE_RETURN_MACRO(GL_NO_ERROR);
        CASE_RETURN_MACRO(GL_INVALID_ENUM);
        CASE_RETURN_MACRO(GL_INVALID_VALUE);
        CASE_RETURN_MACRO(GL_INVALID_OPERATION);
        CASE_RETURN_MACRO(GL_OUT_OF_MEMORY);
        CASE_RETURN_MACRO(GL_STACK_UNDERFLOW);
        CASE_RETURN_MACRO(GL_STACK_OVERFLOW);
#ifdef GL_INVALID_FRAMEBUFFER_OPERATION
        CASE_RETURN_MACRO(GL_INVALID_FRAMEBUFFER_OPERATION);
#endif
        default: break;
    }
#undef CASE_RETURN_MACRO
    return "*UNKNOWN*";
}

bool checkGLError(const char* msg = "")
{
	GLenum gl_error = glGetError();
    if (gl_error != GL_NO_ERROR)
    {
		std::cout << "GL error: " << glErrorToString(gl_error) << " msg: " << msg << "\n";
		return false;
	}
	return true;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

int main(void)
{
    //Simulation parameters:
    const int N = 8192*4;
    const int block_sz = 1024;
    const int n_blocks = N / block_sz;
    const float G = 2e-4f;
    const float dt = 5e-3f;

    //Create window:
	int width = 640;
	int height = 480;

    glfwSetErrorCallback(error_callback);
	if (!glfwInit()){ return -1; }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(width, height, "Simple example", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
	}
    glfwSetKeyCallback(window, key_callback);
    glfwMakeContextCurrent(window);
	glfwSwapInterval(0);

	glewExperimental = GL_TRUE;
	GLenum err = glewInit();
	glGetError();
	if (err != GLEW_OK)
	{
		std::cout << "glewInit failed: " << glewGetErrorString(err);
		return -1;
	}

    glViewport(0, 0, width, height);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

	//verify the openGL version we got:
	{
		int p = glfwGetWindowAttrib(window, GLFW_OPENGL_PROFILE);
		std::string version = glfwGetVersionString();
		std::string opengl_profile = "";
		if      (p == GLFW_OPENGL_COMPAT_PROFILE){ opengl_profile = "OpenGL Compatibility Profile"; }
		else if (p == GLFW_OPENGL_CORE_PROFILE  ){ opengl_profile = "OpenGL Core Profile"; }
		std::cout << "GLFW version: " << version << "\n";
		std::cout << "GLFW OpenGL profile: " << opengl_profile << "\n";

		std::cout << "OpenGL: GL version: " <<  glGetString(GL_VERSION) << "\n";
		std::cout << "OpenGL: GLSL version: " << glGetString(GL_SHADING_LANGUAGE_VERSION) << "\n";
		std::cout << "OpenGL: Vendor: " << glGetString(GL_VENDOR) << "\n";

		std::cout << "GLEW: Glew version: " << glewGetString(GLEW_VERSION) << "\n";
	}
    
    //Generate data for Nbody simulation:
    std::vector<Particle> particles(N);
    std::vector<Velocity> velocities(N);

    std::mt19937 mersenne_engine{43};  // Generates random integers
    std::uniform_real_distribution<float> dist1{-8.5f, 8.5f};
    std::uniform_real_distribution<float> dist2{0.5f, 15.0f};
    std::uniform_real_distribution<float> dist3{-0.5f, 0.5f};
    auto gen = [&]()
    { 
        float disp = dist3(mersenne_engine) > 0.0f ? -10.0f : +10.0f;
        return Particle{dist1(mersenne_engine) + disp, dist1(mersenne_engine) - disp/2, dist1(mersenne_engine), dist2(mersenne_engine)};
    };
    std::generate(particles.begin(), particles.end(), gen);
    auto genv = [&]()
    {
        return Velocity{dist1(mersenne_engine)*0.11f, dist1(mersenne_engine)*0.01f, dist1(mersenne_engine)*0.001f};
    };
    std::generate(velocities.begin(), velocities.end(), genv);
    /*particles[0].m = 20000.0f;
    particles[1].m = 32000.0f;
    particles[2].m = 15000.0f;
    particles[3].m = 18000.0f;*/

	//Compile shaders:
	auto load_and_compile_shader = [](auto shader_type, std::string const& path)->GLuint
	{
		std::basic_string<GLchar> string;

		if(path.size() != 0)
		{
			std::basic_ifstream<GLchar> file(path);
			if(!file.is_open()){ std::cout << "Cannot open shader file: " << path << "\n"; return 0; }
			string = std::basic_string<GLchar>( std::istreambuf_iterator<GLchar>(file), (std::istreambuf_iterator<GLchar>()));
		}
		else
		{
			//string = std::basic_string<GLchar>{ shader_type == GL_VERTEX_SHADER ? vertex_shader_str : fragment_shader_str };
			return 0;
		}
		const GLchar* tmp = string.c_str();

		auto shaderObj = glCreateShader(shader_type);
		if(!checkGLError()){ return 0; }
		
		GLint gl_status = 0;
		glShaderSource(shaderObj, (GLsizei)1, &tmp, NULL);
		glCompileShader(shaderObj);
		glGetShaderiv(shaderObj, GL_COMPILE_STATUS, &gl_status);

		if (!gl_status)
		{
			GLint log_size;
			glGetShaderiv(shaderObj, GL_INFO_LOG_LENGTH, &log_size);
			std::basic_string<GLchar> log(log_size, ' ');
			glGetShaderInfoLog(shaderObj, log_size, NULL, &(*log.begin()));
			std::cout << "Failed to compile shader: " << std::endl << log << std::endl;
		}
		else
		{
			std::cout << "Shader " << path << " compiled successfully\n";
		}

		return shaderObj;
	};

	GLuint vertexShaderObj   = load_and_compile_shader(GL_VERTEX_SHADER,   "vertex.glsl");
	GLuint fragmentShaderObj = load_and_compile_shader(GL_FRAGMENT_SHADER, "fragment.glsl");
	if(!vertexShaderObj && !fragmentShaderObj){ std::cout << "Failed to load and compile shaders\n"; return -1; }

	GLuint glProgram = glCreateProgram();
	{
		glAttachShader(glProgram, vertexShaderObj);
		glAttachShader(glProgram, fragmentShaderObj);
		glLinkProgram(glProgram);

		GLint gl_status = 0;
		glGetProgramiv(glProgram, GL_LINK_STATUS, &gl_status);
		if(!gl_status)
		{
			char temp[256];
			glGetProgramInfoLog(glProgram, 256, 0, temp);
			std::cout << "Failed to link program: " << temp << std::endl;
			glDeleteProgram(glProgram);
		}
		else{ std::cout << "Shaders linked successfully\n"; }

		glUseProgram(glProgram);
		if(!checkGLError()){ return -1; }
	}

    //Create buffers:
    GLuint glbuffer;
    hipGraphicsResource* cuda_glbuffer = nullptr;
	GLuint glvao;
	
	//Create buffer ID
	glGenBuffers(1, &glbuffer);
	if(!checkGLError("glGenBuffers")){ return -1; }

	// Select the GL Buffer as the active one: 
	glBindBuffer(GL_ARRAY_BUFFER, glbuffer);
	if(!checkGLError("glBindBuffer")){ return -1; }

	// Allocate memory for the buffer:
	glBufferData(GL_ARRAY_BUFFER, particles.size() * sizeof(Particle), NULL, GL_STATIC_DRAW);
	if(!checkGLError("glBufferData")){ return -1; }

	// Upload data to the buffer:
	glBufferSubData(GL_ARRAY_BUFFER, 0, particles.size() * sizeof(Particle), particles.data());
	if(!checkGLError("glBufferSubData")){ return -1; }

	// Create and activate the Vertex Array Object
	// these API functions are missing on windows from the CUDA SDK glew, so we load them manually:
	typedef void (*Fnt_GenVertexArrays) (GLsizei n, GLuint *arrays);
	typedef void (*Fnt_BindVertexArray) (GLuint array);
	Fnt_GenVertexArrays pglGenVertexArrays = load_extension_pointer<Fnt_GenVertexArrays>("glGenVertexArrays");
	Fnt_BindVertexArray pglBindVertexArray = load_extension_pointer<Fnt_BindVertexArray>("glBindVertexArray");

	pglGenVertexArrays(1, &glvao); if(!checkGLError("glGenVertexArrays")){ return -1; }
	pglBindVertexArray(glvao);	   if(!checkGLError("glBindVertexArray")){ return -1; }

	// Register buffers into the VAO
	glBindBuffer(GL_ARRAY_BUFFER, glbuffer);
	if(!checkGLError("glBindBuffer(geo)")){ return -1; }
	glVertexAttribPointer(0, 4, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid *)0);
	if(!checkGLError("glVertexAttribPointer(geo)")){ return -1; }
	
	// register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_glbuffer, glbuffer, cudaGraphicsMapFlagsWriteDiscard));
	
	glEnableVertexAttribArray(0);
	if(!checkGLError("glEnableVertexAttribArray(0)")){ return -1; }
	
	//Cuda source arrays:
    float4* cuda_particles = nullptr;
    float3* cuda_velocities = nullptr;
	auto cuerr = hipMalloc( (void**)&cuda_particles, particles.size()*sizeof(Particle) );
    if( cuerr != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(cuerr) << "\n"; return -1; }
    
    cuerr = hipMalloc( (void**)&cuda_velocities, velocities.size()*sizeof(Velocity) );
	if( cuerr != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(cuerr) << "\n"; return -1; }
	
	cuerr = hipMemcpy( cuda_particles, particles.data(), particles.size()*sizeof(Particle), hipMemcpyHostToDevice );
    if( cuerr != hipSuccess){ std::cout << "Error copying memory to device: " << hipGetErrorString(cuerr) << "\n"; return -1; }

    cuerr = hipMemcpy( cuda_velocities, velocities.data(), velocities.size()*sizeof(Velocity), hipMemcpyHostToDevice );
    if( cuerr != hipSuccess){ std::cout << "Error copying memory to device: " << hipGetErrorString(cuerr) << "\n"; return -1; }


	std::cout << "Entering render loop\n";
    while (!glfwWindowShouldClose(window))
    {
		// map OpenGL buffer object for writing from CUDA
		float4* ptr;
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_glbuffer, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes, cuda_glbuffer));

		// execute the kernel
		dim3 dimGrid( n_blocks );
        dim3 dimBlock( block_sz );
		gpu_nbody_opt<<<dimGrid, dimBlock>>>(cuda_velocities, ptr, G, dt);

		// unmap buffer object
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_glbuffer, 0));

		glClearColor(0, 0, 0, 1);
		glClear(GL_COLOR_BUFFER_BIT);
        
        pglBindVertexArray(glvao);
		glDrawArrays(GL_POINTS, 0, N);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
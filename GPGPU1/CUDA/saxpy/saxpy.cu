
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <iostream>

__global__ void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        y[i] = a*x[i] + y[i];
    }
}

int main()
{
    const float A = 100.0f;
	std::vector<float> X{0.f, 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f, 8.f, 9.f};
	std::vector<float> Y{0.f, 2.f, 4.f, 6.f, 8.f, 10.f, 12.f, 14.f, 16.f, 18.f};
	std::vector<float> R(X.size());

	size_t sz = X.size();
	float* devX = nullptr;
	float* devY = nullptr;

	hipError_t err = hipSuccess;
	err = hipMalloc( (void**)&devX, sz*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory (X): " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&devY, sz*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory (Y): " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMemcpy( devX, X.data(), sz*sizeof(float), hipMemcpyHostToDevice );
	if( err != hipSuccess){ std::cout << "Error copying memory to device (X): " << hipGetErrorString(err) << "\n"; return -1; }
    
    err = hipMemcpy( devY, Y.data(), sz*sizeof(float), hipMemcpyHostToDevice );
	if( err != hipSuccess){ std::cout << "Error copying memory to device (Y): " << hipGetErrorString(err) << "\n"; return -1; }
    
	dim3 dimGrid( 1 );
	dim3 dimBlock( sz );
	saxpy<<<dimGrid, dimBlock>>>((int)sz, A, devX, devY);

	err = hipGetLastError();
	if (err != hipSuccess){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipMemcpy( R.data(), devY, sz*sizeof(float), hipMemcpyDeviceToHost );
	if( err != hipSuccess){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( devX );
	if( err != hipSuccess){ std::cout << "Error freeing allocation (X): " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( devY );
	if( err != hipSuccess){ std::cout << "Error freeing allocation (Y): " << hipGetErrorString(err) << "\n"; return -1; }

	for( auto r : R )
	{
		std::cout << r << "\n";
	}

	std::transform(X.begin(), X.end(), Y.begin(), Y.begin(), [a = A](float x, float y){ return a*x + y; });

	if(std::equal(R.begin(), R.end(), Y.begin()))
	{
		std::cout << "Success\n";
	}
	else{ std::cout << "Mismatch between CPU and GPU results.\n"; }
	
	return 0;
}

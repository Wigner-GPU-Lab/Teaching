#include <hip/hip_runtime.h>

#include <stdio.h>  // printf

void checkErr(hipError_t err, const char * name)
{
    if (err != hipSuccess)
    {
        printf("ERROR: %s (%i)\n", name, err);
        exit( err );
    }
}

int main()
{
    hipError_t err = hipSuccess;
    int numDevices = 0;

    err = hipGetDeviceCount(&numDevices);
    checkErr(err, "cudaGetDeviceCount()");

    if (numDevices == 0)
    {
        printf("No CUDA devices detected.\n");
        exit( -1 );
    }
    printf("Found %u device(s)\n", numDevices);
    fflush(NULL);

    for (int i = 0; i < numDevices; ++i)
    {
        hipDeviceProp_t deviceProps;
        err = hipGetDeviceProperties(&deviceProps, i);
        checkErr(err, "cudaGetDeviceProperties()");

        printf("\t%s\n", deviceProps.name);
    }

    return 0;
}

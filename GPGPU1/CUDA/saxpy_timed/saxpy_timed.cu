
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <chrono>

__global__ void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        y[i] = a*x[i] + y[i];
    }
}

int main()
{
    size_t sz = 1 << 24;
    std::cout << "Data size is: " << sz << "\n";
    const float A = 2.0f;

	std::vector<float> X(sz);
	std::vector<float> Y(sz);
    std::vector<float> R(sz);
    
    std::generate(X.begin(), X.end(), [d=0.f, dd=+1.0f/sz]()mutable{ d += dd; return d; });
    std::generate(Y.begin(), Y.end(), [d=1.f, dd=-1.0f/sz]()mutable{ d += dd; return d; });
	
	float* devX = nullptr;
	float* devY = nullptr;

	hipError_t err = hipSuccess;
	err = hipMalloc( (void**)&devX, sz*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory (X): " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&devY, sz*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory (Y): " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMemcpy( devX, X.data(), sz*sizeof(float), hipMemcpyHostToDevice );
	if( err != hipSuccess){ std::cout << "Error copying memory to device (X): " << hipGetErrorString(err) << "\n"; return -1; }
    
    err = hipMemcpy( devY, Y.data(), sz*sizeof(float), hipMemcpyHostToDevice );
	if( err != hipSuccess){ std::cout << "Error copying memory to device (Y): " << hipGetErrorString(err) << "\n"; return -1; }
    
    hipEvent_t evt[2];
    for(auto& e : evt)
    {
        err = hipEventCreate(&e);
        if( err != hipSuccess){ std::cout << "Error creating event: " << hipGetErrorString(err) << "\n"; return -1; }
    }

	dim3 dimGrid( sz/512 );
    dim3 dimBlock( 512 );
    
    err = hipEventRecord(evt[0]);
    if( err != hipSuccess){ std::cout << "Error recording event (0): " << hipGetErrorString(err) << "\n"; return -1; }
    
    saxpy<<<dimGrid, dimBlock>>>((int)sz, A, devX, devY);
    
    err = hipGetLastError();
	if (err != hipSuccess){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
    
    err = hipEventRecord(evt[1]);
    if( err != hipSuccess){ std::cout << "Error recording event (1): " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipMemcpy( R.data(), devY, sz*sizeof(float), hipMemcpyDeviceToHost );
	if( err != hipSuccess){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( devX );
	if( err != hipSuccess){ std::cout << "Error freeing allocation (X): " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( devY );
	if( err != hipSuccess){ std::cout << "Error freeing allocation (Y): " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipEventSynchronize(evt[1]);
    if( err != hipSuccess){ std::cout << "Error during synchronize with event: " << hipGetErrorString(err) << "\n"; return -1; }

    float dt = 0.0f;//milliseconds
    err = hipEventElapsedTime(&dt, evt[0], evt[1]);
    if( err != hipSuccess){ std::cout << "Error getting event elapsed time: " << hipGetErrorString(err) << "\n"; return -1; }

    for(auto& e : evt)
    {
        err = hipEventDestroy(e);
        if( err != hipSuccess){ std::cout << "Error destroying event: " << hipGetErrorString(err) << "\n"; return -1; }
    }

    // skip printing:
	/*for( auto r : R )
	{
		std::cout << r << "\n";
	}*/

    auto t0 = std::chrono::high_resolution_clock::now();
	std::transform(X.begin(), X.end(), Y.begin(), Y.begin(), [a = A](float x, float y){ return a*x + y; });
    auto t1 = std::chrono::high_resolution_clock::now();
    
	if(std::equal(R.begin(), R.end(), Y.begin()))
	{
		std::cout << "Success\n";
	}
    else{ std::cout << "Mismatch between CPU and GPU results.\n"; }
    
    std::cout << "CPU computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count()/1000.0f << " ms\n";
    std::cout << "GPU computation took: " << dt << " ms.\n";

	return 0;
}

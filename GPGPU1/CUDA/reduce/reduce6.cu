
#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>

__device__ void warpReduce(volatile float* tmp, int tid)
{
    tmp[tid] += tmp[tid + 32];
    tmp[tid] += tmp[tid + 16];
    tmp[tid] += tmp[tid + 8];
    tmp[tid] += tmp[tid + 4];
    tmp[tid] += tmp[tid + 2];
    tmp[tid] += tmp[tid + 1];
}

__global__ void reduce(float* dst, float* src, int n) 
{
    extern __shared__ float tmp[];
    static const unsigned int blockSize = 512;

    // each thread loads two elements from global, adds them
    // and stores the result to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    tmp[tid] = 0;
    while(i < n){ tmp[tid] += src[i] + src[i+blockSize]; i += gridSize; }
    __syncthreads();
    
    // do reduction in shared mem, no loop, it was unrolled
    if(tid < 256){ tmp[tid] += tmp[tid + 256]; } __syncthreads();
    if(tid < 128){ tmp[tid] += tmp[tid + 128]; } __syncthreads();
    if(tid <  64){ tmp[tid] += tmp[tid +  64]; } __syncthreads();
    if(tid <  32){ warpReduce(tmp, tid); }
    
    // write result for this block to global mem
    if(tid == 0){ dst[blockIdx.x] = tmp[0]; }
}

int main()
{
    //const int divisor1 = reduce_version <= 3 ? 1 : 2;// = 1, but = 2 for kernels 4, and above
    //const int divisor2 = reduce_version <= 5 ? divisor1 : 1;// = divisor1, but = 1 for kernel6 and 7
    
    const size_t extra = 2;
    const size_t sz = 512*512*extra;
    const size_t block_sz = 512;
    const int    n_blocks = sz / extra / block_sz;

	std::vector<float> A(sz);
	std::vector<float> B(n_blocks);

    std::mt19937 mersenne_engine{42};  // Generates random integers
    std::uniform_real_distribution<float> dist{-0.1f, 0.1f};

    auto gen = [&dist, &mersenne_engine](){ return dist(mersenne_engine); };
    generate(A.begin(), A.end(), gen);
	
	float* src = nullptr;
    float* dst = nullptr;
    float* res = nullptr;
    float  gpu_sum = 0.0f;

    hipEvent_t evt[4];
    for(auto& e : evt){ hipEventCreate(&e); }

	hipError_t err = hipSuccess;
	err = hipMalloc( (void**)&src, sz*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMalloc( (void**)&dst, n_blocks*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
    
    err = hipMalloc( (void**)&res, 1*sizeof(float) );
	if( err != hipSuccess){ std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n"; return -1; }
	
	err = hipMemcpy( src, A.data(), sz*sizeof(float), hipMemcpyHostToDevice );
	if( err != hipSuccess){ std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }
    
    {
        dim3 dimGrid( n_blocks/2, 1 );//<<<<<<<<<<divide by 2!
        dim3 dimBlock( block_sz, 1 );
        size_t shared_mem_size = block_sz*sizeof(float);
        hipEventRecord(evt[0]);
        reduce<<<dimGrid, dimBlock, shared_mem_size>>>(dst, src, dimGrid.x*dimBlock.x*2*extra);//<<<<<mul by 2
        err = hipGetLastError();
	    if (err != hipSuccess){ std::cout << "CUDA error in kernel call 1: " << hipGetErrorString(err) << "\n"; return -1; }
        hipEventRecord(evt[1]);
    }
	
    {
        dim3 dimGrid( 1, 1 );
        dim3 dimBlock( block_sz, 1 );//<<<<<<<div by 1!
        size_t shared_mem_size = block_sz*sizeof(float);
        hipEventRecord(evt[2]);
        reduce<<<dimGrid, dimBlock, shared_mem_size>>>(res, dst, dimGrid.x*dimBlock.x);
        err = hipGetLastError();
	    if (err != hipSuccess){ std::cout << "CUDA error in kernel call 2: " << hipGetErrorString(err) << "\n"; return -1; }
        hipEventRecord(evt[3]);
    }

	err = hipMemcpy( &gpu_sum, res, sizeof(float), hipMemcpyDeviceToHost );
	if( err != hipSuccess){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( src );
	if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFree( dst );
	if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFree( res );
	if( err != hipSuccess){ std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n"; return -1; }

    hipEventSynchronize(evt[3]);
    float dt1 = 0.0f, dt2 = 0.0f;//milliseconds
    hipEventElapsedTime(&dt1, evt[0], evt[1]);
    hipEventElapsedTime(&dt2, evt[2], evt[3]);

    for(auto& e : evt){ hipEventDestroy(e); }

    auto t0 = std::chrono::high_resolution_clock::now();
    float cpu_sum = std::accumulate(A.begin(), A.end(), 0.0f);
    auto t1 = std::chrono::high_resolution_clock::now();

    std::cout.precision(10);
    std::cout << "cpu_sum = " << cpu_sum << "\n";
    std::cout << "gpu_sum = " << gpu_sum << "\n";

    float rel_err = std::abs((cpu_sum - gpu_sum) / cpu_sum);
    std::cout << "Relative error is: " << rel_err << "\n";
	if( rel_err < 2e-4 )
	{
		std::cout << "Success.\n";
	}
	else
	{
		std::cout << "Mismatch in CPU and GPU result.\n";
    }
    
    std::cout << "CPU Computation took: " << std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count()/1000.0f << " ms\n";
    std::cout << "GPU Computation took: " << dt1 << " + " << dt2 << " = " << dt1+dt2 << " ms.\n";
	return 0;
}
